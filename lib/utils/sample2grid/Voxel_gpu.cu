#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <myGridSampler.cuh>
#include <vector>
#include <stdio.h>

//using namespace std;
namespace{

template <typename scalar_t>
__global__ void voxel_2d_kernel(
          const at::PackedTensorAccessor64<scalar_t,3,at::RestrictPtrTraits> input,
          const at::PackedTensorAccessor64<scalar_t,3,at::RestrictPtrTraits> grid,
           at::PackedTensorAccessor64<scalar_t,4,at::RestrictPtrTraits> output,
           at::PackedTensorAccessor64<int,3,at::RestrictPtrTraits> output_count)
{
    // input (N,C,H)
    // grid (N,H,Coor)
    // output (N,C, H, W)
    // output_count (N,H,W)
    int C = input.size(1);
    int input_H=input.size(2);

    int out_H = output.size(2);
    int out_W = output.size(3);

    int grid_H=grid.size(1);
    int grid_Coor=grid.size(2);

        //batch index
      const int n = blockIdx.y;
      // column index
      const int h = blockIdx.x * blockDim.x + threadIdx.x;
      if(h < input_H){
        // get the corresponding input x, y co-ordinates from grid
      scalar_t ix = grid[n][h][0];
      scalar_t iy = grid[n][h][1];

      ix = grid_sampler_compute_source_index(ix, out_W);
      iy = grid_sampler_compute_source_index(iy, out_H);

        int ix_nearest = static_cast<int>(::round(ix));
        int iy_nearest = static_cast<int>(::round(iy));

        // assign nearest neighor pixel value to output pixel
        if (within_bounds_2d(iy_nearest, ix_nearest, out_H, out_W)) {
            atomicAdd((int* )&(output_count[n][iy_nearest][ix_nearest]), int(1));
//            safe_add_2d(count_ptr, iy_nearest, ix_nearest, out_ct_sH, out_ct_sW, out_H, out_W, 1);
            for (int c = 0; c < C; ++c) {
              // calculate and set grad_input
              atomicAdd((scalar_t* )&(output[n][c][iy_nearest][ix_nearest]),input[n][c][h]);
            }
        }
      }
}

template <typename scalar_t>
__global__ void voxel_2d_normal_kernel(
           at::PackedTensorAccessor64<scalar_t,4,at::RestrictPtrTraits> output,
           const at::PackedTensorAccessor64<int,3,at::RestrictPtrTraits> output_count)
{
    // output (N,C, H, W)
    // output_count (N,H,W)
    int C = output.size(1);
    int out_H = output.size(2);
    int out_W = output.size(3);


        //batch index
      const int n = blockIdx.y;
      // column index
      const int hw = blockIdx.x * blockDim.x + threadIdx.x;
      const int h=hw/out_W;
      const int w=hw -h*out_W;
      if(h < out_H &&w < out_W){
        // get the corresponding input x, y coordinates from grid
        // assign nearest neighbor pixel value to output pixel
        int ct=output_count[n][h][w];
        if(ct>0){
            for (int c=0;c<C;c++){
                output[n][c][h][w]/=ct;
            }
        }
      }
}

template <typename scalar_t>
__global__ void voxel_2d_backward_kernel(
  const at::PackedTensorAccessor64<scalar_t,3,at::RestrictPtrTraits> grid,
  const at::PackedTensorAccessor64<int,3,at::RestrictPtrTraits> output_count,
  const at::PackedTensorAccessor64<scalar_t,4,at::RestrictPtrTraits> grad_output,
  at::PackedTensorAccessor64<scalar_t,3,at::RestrictPtrTraits> grad_input)
{

    // grid (N,H,Coor)
    // output_count (N, H, W)
    // grad_output (N,C,H,W)
    // grad_input (N,C,H2)

    int C = grad_output.size(1);
    int gInp_H = grad_input.size(2);

    int grid_H = grid.size(1);

    int out_H=output_count.size(1);
    int out_W=output_count.size(2);

        //batch index
      const int n = blockIdx.y;
      // column index
      const int h = blockIdx.x * blockDim.x + threadIdx.x;
      if(h < gInp_H){
            // get the corresponding input x, y co-ordinates from grid
          scalar_t ix = grid[n][h][0];
          scalar_t iy = grid[n][h][1];

          ix = grid_sampler_compute_source_index(ix, out_W);
          iy = grid_sampler_compute_source_index(iy, out_H);


            int ix_nearest = static_cast<int>(::round(ix));
            int iy_nearest = static_cast<int>(::round(iy));

            // assign nearest neighor pixel value to output pixel
            auto ct= output_count[n][iy_nearest][ix_nearest];
            if(ct<=0 || !within_bounds_2d(iy_nearest, ix_nearest, out_H, out_W)){
                //TODO check here
                for (int c = 0; c < C; ++c) {
                    grad_input[n][c][h] = static_cast<scalar_t>(0);
                }
            }else{
                for (int c = 0; c < C; ++c) {
//                    printf('%f',static_cast<float>(grad_output[n][c][iy_nearest][ix_nearest]/ct));
                    grad_input[n][c][h] = grad_output[n][c][iy_nearest][ix_nearest]/(float)ct;
                }
            }
      }

}

}//namespace

// No shape checking needed here. See # NOTE [ grid_sampler Native Functions ].
std::tuple<torch::Tensor, torch::Tensor>
grid_voxel_2d_cuda_forward(const torch::Tensor& input, const torch::Tensor& grid, torch::Tensor& output, torch::Tensor& output_count) {
  const auto N = grid.size(0);
  const auto H = grid.size(1);

  const int threads=1024;
  const dim3 blocks((H+threads-1)/threads, N);

//    AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "grid_voxel_2d_cuda", ([&] {
      voxel_2d_kernel<float>
        <<<blocks,threads>>>(
          input.packed_accessor64<float,3,torch::RestrictPtrTraits>(),
          grid.packed_accessor64<float,3,torch::RestrictPtrTraits>(),
          output.packed_accessor64<float,4,torch::RestrictPtrTraits>(),
          output_count.packed_accessor64<int,3,torch::RestrictPtrTraits>());
//    }));
         const auto out_H=output.size(2);
         const auto out_W=output.size(3);
        dim3 blocks2((out_H*out_W+threads-1)/threads, N);

       voxel_2d_normal_kernel<float>
       <<<blocks2,threads>>>(
          output.packed_accessor64<float,4,torch::RestrictPtrTraits>(),
          output_count.packed_accessor64<int,3,torch::RestrictPtrTraits>()
       );

  return std::make_tuple(output,output_count);
};

// No shape checking needed here. See # NOTE [ grid_sampler Native Functions ].
torch::Tensor grid_voxel_2d_cuda_backward(const torch::Tensor& grid, const torch::Tensor& output_count,
                            const torch::Tensor& grad_output,torch::Tensor& grad_input) {
  const auto N = grid.size(0);
  const auto H = grid.size(1);

  const int threads=1024;
  const dim3 blocks((H+threads-1)/threads, N);


//    AT_DISPATCH_FLOATING_TYPES(output_count.scalar_type(), "grid_voxel_2d_backward_cuda", ([&] {
      voxel_2d_backward_kernel<float>
        <<<blocks,threads>>>(
          grid.packed_accessor64<float,3,torch::RestrictPtrTraits>(),
          output_count.packed_accessor64<int,3,torch::RestrictPtrTraits>(),
          grad_output.packed_accessor64<float,4,torch::RestrictPtrTraits>(),
          grad_input.packed_accessor64<float,3,torch::RestrictPtrTraits>()
          );

//    }
//    ));
  return grad_input;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  m.def("grid_voxel_2d_cuda_forward", &grid_voxel_2d_cuda_forward, "grid_voxel_2d_cuda");
  m.def("grid_voxel_2d_cuda_backward", &grid_voxel_2d_cuda_backward, "grid_voxel_2d_backward_cuda");
}