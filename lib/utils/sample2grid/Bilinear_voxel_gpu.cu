#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <myGridSampler.cuh>
#include <vector>
#include <stdio.h>

//using namespace std;
namespace{

template <typename scalar_t>
__global__ void bilinear_voxel_2d_kernel(
    const at::PackedTensorAccessor64<scalar_t, 3, at::RestrictPtrTraits> input,
    const at::PackedTensorAccessor64<scalar_t, 3, at::RestrictPtrTraits> grid,
    at::PackedTensorAccessor64<scalar_t, 4, at::RestrictPtrTraits> output,
    at::PackedTensorAccessor64<float, 3, at::RestrictPtrTraits> output_bilinear_count
)
{
    // input (N,C,H)
    // grid (N,H,Coor)
    // output (N,C, H, W)
    // output_bilinear_count (N,H,W)
    int C = input.size(1);
    int input_H = input.size(2);

    int out_H = output.size(2);
    int out_W = output.size(3);

    int grid_H=grid.size(1);
    int grid_Coor=grid.size(2);

        //batch index
    const int n = blockIdx.y;
    // column index
    const int h = blockIdx.x * blockDim.x + threadIdx.x;
    if(h < input_H){
        // get the corresponding input x, y co-ordinates from grid
      float ix = static_cast<float>(grid[n][h][0]);
      float iy = static_cast<float>(grid[n][h][1]);

        ix = grid_sampler_compute_source_index(ix, out_W);
        iy = grid_sampler_compute_source_index(iy, out_H);
        int ix0 = static_cast<int>(::floor(ix));
//        int ix1 = ::ceil(ix);
        int iy0 = static_cast<int>(::floor(iy));
//        int iy1 = ::ceil(iy);

//        float ix_rest = static_cast<float>(ix_nearest - ix);
//        float iy_rest = static_cast<float>(iy_nearest - iy);
        // hand-craft 4 points
        float weight =0;
        for (int i = ix0; i <= ix0+1; ++i){
            for (int j = iy0; j <= iy0+1; ++j){
                // assign nearest neighor pixel value to output pixel
                if (within_bounds_2d(j, i, out_H, out_W)) {
                    // bilinear: exp(-(ix**2+iy**2))

                    weight = (1.-::fabs(ix-i))*(1. - ::fabs(iy-j));
//# if __CUDA_ARCH__>=200
//    printf("weight, %f \n", weight);
//#endif
//                       float weight = 1.;
                //            atomicAdd((int* )&(output_count[n][iy_nearest][ix_nearest]), int(1));
                    atomicAdd((float* )&(output_bilinear_count[n][j][i]), 1.);
                //            safe_add_2d(count_ptr, iy_nearest, ix_nearest, out_ct_sH, out_ct_sW, out_H, out_W, 1);
                    for (int c = 0; c < C; ++c) {
                      // calculate and set grad_input
                      atomicAdd((scalar_t* )&(output[n][c][j][i]),weight*input[n][c][h]);
                    }
                }
            }
        }

    }
}

template <typename scalar_t>
__global__ void bilinear_voxel_2d_normal_kernel(
           at::PackedTensorAccessor64<scalar_t,4,at::RestrictPtrTraits> output ,
           const at::PackedTensorAccessor64<float, 3, at::RestrictPtrTraits> output_bilinear_count)
{
    // output (N,C, H, W)
    // output_count (N,H,W)
    int C = output.size(1);
    int out_H = output.size(2);
    int out_W = output.size(3);


        //batch index
      const int n = blockIdx.y;
      // column index
      const int hw = blockIdx.x * blockDim.x + threadIdx.x;
      const int h=hw/out_W;
      const int w=hw -h*out_W;
      if(h < out_H &&w < out_W){
        // get the corresponding input x, y co-ordinates from grid
        // assign nearest neighor pixel value to output pixel
        float bilinear_ct=output_bilinear_count[n][h][w];
        if(bilinear_ct>0){
            for (int c=0;c<C;c++){
                output[n][c][h][w]/=bilinear_ct;
            }
        }
      }
}

template <typename scalar_t>
__global__ void bilinear_voxel_2d_backward_kernel(
  const at::PackedTensorAccessor64<scalar_t,3,at::RestrictPtrTraits> grid,
  const at::PackedTensorAccessor64<float,3,at::RestrictPtrTraits> output_bilinear_count,
  const at::PackedTensorAccessor64<scalar_t,4,at::RestrictPtrTraits> grad_output,
  at::PackedTensorAccessor64<scalar_t,3,at::RestrictPtrTraits> grad_input)
{

    // grid (N,H,Coor)
    // output_count (N, H, W)
    // grad_output (N,C,H,W)
    // grad_input (N,C,H2)

    int C = grad_output.size(1);
    int gInp_H = grad_input.size(2);

    int grid_H = grid.size(1);

    int out_H=output_bilinear_count.size(1);
    int out_W=output_bilinear_count.size(2);

        //batch index
      const int n = blockIdx.y;
      // column index
      const int h = blockIdx.x * blockDim.x + threadIdx.x;
      if(h < gInp_H){
            // get the corresponding input x, y co-ordinates from grid
          float ix = static_cast<float>(grid[n][h][0]);
          float iy = static_cast<float>(grid[n][h][1]);

          ix = grid_sampler_compute_source_index(ix, out_W);
          iy = grid_sampler_compute_source_index(iy, out_H);


        int ix0 = static_cast<int>(::floor(ix));
//        int ix1 = ::ceil(ix);
        int iy0 = static_cast<int>(::floor(iy));
        float weight =0.;
        // assign nearest neighor pixel value to output pixel
        for (int i = ix0; i <= ix0; ++i){
            for (int j = iy0; j <= iy0; ++j){
                auto ct= output_bilinear_count[n][j][i];
                if(ct<=0 || !within_bounds_2d(j, i, out_H, out_W)){
                    //TODO check here
                    for (int c = 0; c < C; ++c) {
                        grad_input[n][c][h] = static_cast<scalar_t>(0);
                    }
                }else{
                    for (int c = 0; c < C; ++c) {
                         weight  = (1.-::fabs(ix-i))*(1. - ::fabs(iy-j));
//                          float weight = 1.;
                //                    printf('%f',static_cast<float>(grad_output[n][c][iy_nearest][ix_nearest]/ct));
                        grad_input[n][c][h] = grad_output[n][c][j][i]*weight/ct;
                    }
                }
            }
        }
      }
}

}//namespace

// No shape checking needed here. See # NOTE [ grid_sampler Native Functions ].
std::tuple<torch::Tensor, torch::Tensor>
bilinear_grid_voxel_2d_cuda_forward(const torch::Tensor& input, const torch::Tensor& grid, torch::Tensor& output, torch::Tensor& output_bilinear_count) {
  const auto N = grid.size(0);
  const auto H = grid.size(1);

  const int threads=1024;
  const dim3 blocks((H+threads-1)/threads, N);

//    AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "grid_voxel_2d_cuda", ([&] {
      bilinear_voxel_2d_kernel<float>
        <<<blocks,threads>>>(
          input.packed_accessor64<float,3,torch::RestrictPtrTraits>(),
          grid.packed_accessor64<float,3,torch::RestrictPtrTraits>(),
          output.packed_accessor64<float,4,torch::RestrictPtrTraits>(),
          output_bilinear_count.packed_accessor64<float,3,torch::RestrictPtrTraits>());
//    }));
         const auto out_H=output.size(2);
         const auto out_W=output.size(3);
        dim3 blocks2((out_H*out_W+threads-1)/threads, N);

       bilinear_voxel_2d_normal_kernel<float>
       <<<blocks2,threads>>>(
          output.packed_accessor64<float,4,torch::RestrictPtrTraits>(),
          output_bilinear_count.packed_accessor64<float,3,torch::RestrictPtrTraits>()
       );

  return std::make_tuple(output,output_bilinear_count);
};

// No shape checking needed here. See # NOTE [ grid_sampler Native Functions ].
torch::Tensor bilinear_grid_voxel_2d_cuda_backward(const torch::Tensor& grid, const torch::Tensor& output_bilinear_count,
                            const torch::Tensor& grad_output,torch::Tensor& grad_input) {
  const auto N = grid.size(0);
  const auto H = grid.size(1);

  const int threads=1024;
  const dim3 blocks((H+threads-1)/threads, N);


//    AT_DISPATCH_FLOATING_TYPES(output_bilinear_count.scalar_type(), "grid_voxel_2d_backward_cuda", ([&] {
      bilinear_voxel_2d_backward_kernel<float>
        <<<blocks,threads>>>(
              grid.packed_accessor64<float,3,torch::RestrictPtrTraits>(),
              output_bilinear_count.packed_accessor64<float,3,torch::RestrictPtrTraits>(),
              grad_output.packed_accessor64<float,4,torch::RestrictPtrTraits>(),
              grad_input.packed_accessor64<float,3,torch::RestrictPtrTraits>()
          );

//    }
//    ));
  return grad_input;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  m.def("bilinear_grid_voxel_2d_cuda_forward", &bilinear_grid_voxel_2d_cuda_forward, "bilinear_grid_voxel_2d_cuda_forward");
  m.def("bilinear_grid_voxel_2d_cuda_backward", &bilinear_grid_voxel_2d_cuda_backward, "bilinear_grid_voxel_2d_cuda_backward");
}